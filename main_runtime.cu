#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "kernels.cuh"
#include <iomanip>

int main() {
    // --- Initialisation (identique à avant) ---
    __hip_bfloat16 *d_A, *d_B; float* d_out;
    const int matrix_size = BLOCK_SIZE * BLOCK_SIZE;
    const int output_size = BLOCK_SIZE;
    CUDA_CHECK(hipMalloc(&d_A, matrix_size * sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&d_B, matrix_size * sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&d_out, output_size * sizeof(float)));
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(32, 1, 1);
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms;

    // --- Mesure Kernel 1 ---
    CUDA_CHECK(hipEventRecord(start));
    Frag_standard_baseline<<<gridDim, blockDim>>>(d_A, d_B, d_out);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    float latency_baseline = ms;

    // --- Mesure Kernel 2 ---
    CUDA_CHECK(hipEventRecord(start));
    Frag_swapped<<<gridDim, blockDim>>>(d_A, d_B, d_out);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    float latency_in_register = ms;

    // --- Impression du résultat pour le script bash ---
    std::cout << std::fixed << std::setprecision(8) << latency_baseline << " " << latency_in_register << std::endl;

    // --- Nettoyage ---
    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_out));
    CUDA_CHECK(hipEventDestroy(start)); CUDA_CHECK(hipEventDestroy(stop));
    return 0;
}
