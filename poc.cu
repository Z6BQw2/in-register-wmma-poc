#include <iostream>
#include <vector>
#include <random>
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <mma.h>

#define BLOCK_SIZE 16

#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

using namespace nvcuda;

__global__ void Frag_standard_baseline(const __hip_bfloat16* A, const __hip_bfloat16* B, float* out) {
    __shared__ float sD[BLOCK_SIZE][BLOCK_SIZE];

    wmma::fragment<wmma::matrix_a, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, __hip_bfloat16, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, __hip_bfloat16, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, float> acc_frag;
    
    wmma::load_matrix_sync(a_frag, A, BLOCK_SIZE);
    wmma::load_matrix_sync(b_frag, B, BLOCK_SIZE);
    wmma::fill_fragment(acc_frag, 0.0f);
    wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

    wmma::store_matrix_sync(&sD[0][0], acc_frag, BLOCK_SIZE, wmma::mem_row_major);
    __syncthreads();

    if (threadIdx.x < BLOCK_SIZE) {
        int my_row = threadIdx.x;
        float row_max = -INFINITY;
        for (int i = 0; i < BLOCK_SIZE; i++) {
            row_max = fmaxf(row_max, sD[my_row][i]);
        }
        out[my_row] = row_max;
    }
}

__global__ void Frag_swapped(const __hip_bfloat16* A, const __hip_bfloat16* B, float* out) {
    wmma::fragment<wmma::matrix_a, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, __hip_bfloat16, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, __hip_bfloat16, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE, float> acc_frag;

    wmma::load_matrix_sync(a_frag, A, BLOCK_SIZE);
    wmma::load_matrix_sync(b_frag, B, BLOCK_SIZE);
    wmma::fill_fragment(acc_frag, 0.0f);
    wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

    float temp_f = acc_frag.x[2];
    acc_frag.x[2] = acc_frag.x[4];
    acc_frag.x[4] = temp_f;
    temp_f = acc_frag.x[3];
    acc_frag.x[3] = acc_frag.x[5];
    acc_frag.x[5] = temp_f;

    float max_A = fmaxf(fmaxf(acc_frag.x[0], acc_frag.x[1]), fmaxf(acc_frag.x[2], acc_frag.x[3]));
    float max_B = fmaxf(fmaxf(acc_frag.x[4], acc_frag.x[5]), fmaxf(acc_frag.x[6], acc_frag.x[7]));

    unsigned int mask = 0xF << ((threadIdx.x / 4) * 4);
    max_A = fmaxf(max_A, __shfl_xor_sync(mask, max_A, 1));
    max_A = fmaxf(max_A, __shfl_xor_sync(mask, max_A, 2));
    max_B = fmaxf(max_B, __shfl_xor_sync(mask, max_B, 1));
    max_B = fmaxf(max_B, __shfl_xor_sync(mask, max_B, 2));

    if(threadIdx.x % 4 == 0){
        int row_group = threadIdx.x / 4;
        
        int row_idx_A = row_group;
        int row_idx_B = row_group + 8;

        out[row_idx_A] = max_A;
        out[row_idx_B] = max_B;
    }
}

// Ref CPU
void cpu_reference(const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& ref_out) {
    std::vector<float> C(BLOCK_SIZE * BLOCK_SIZE);
    for (int i = 0; i < BLOCK_SIZE; ++i) {
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < BLOCK_SIZE; ++k) {
                sum += A[i * BLOCK_SIZE + k] * B[j * BLOCK_SIZE + k];
            }
            C[i * BLOCK_SIZE + j] = sum;
        }
    }

    ref_out.resize(BLOCK_SIZE);
    for (int i = 0; i < BLOCK_SIZE; ++i) {
        float row_max = -INFINITY;
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            row_max = fmaxf(row_max, C[i * BLOCK_SIZE + j]);
        }
        ref_out[i] = row_max;
    }
}

int main() {
    int deviceId;
    hipDeviceProp_t props;
    CUDA_CHECK(hipGetDevice(&deviceId));
    CUDA_CHECK(hipGetDeviceProperties(&props, deviceId));
    std::cout << "Utilisation du GPU: " << props.name << std::endl;

    const int matrix_size = BLOCK_SIZE * BLOCK_SIZE;
    std::vector<float> h_A(matrix_size);
    std::vector<float> h_B(matrix_size);
    std::vector<__hip_bfloat16> h_A_bf16(matrix_size);
    std::vector<__hip_bfloat16> h_B_bf16(matrix_size);

    std::mt19937 gen(1337); // Seed pour la reproductibilité
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for(int i = 0; i < matrix_size; ++i) {
        h_A[i] = dis(gen);
        h_B[i] = dis(gen);
        h_A_bf16[i] = __float2bfloat16(h_A[i]);
        h_B_bf16[i] = __float2bfloat16(h_B[i]);
    }
    
    __hip_bfloat16 *d_A, *d_B;
    float* d_out;
    const int output_size = BLOCK_SIZE;
    std::vector<float> h_out(output_size);

    CUDA_CHECK(hipMalloc(&d_A, matrix_size * sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&d_B, matrix_size * sizeof(__hip_bfloat16)));
    CUDA_CHECK(hipMalloc(&d_out, output_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A_bf16.data(), matrix_size * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B_bf16.data(), matrix_size * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));

    // Ref CPU
    std::vector<float> ref_out;
    cpu_reference(h_A, h_B, ref_out);

    dim3 gridDim(1, 1, 1);
    dim3 blockDim(32, 1, 1);
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    // Benchmark Baseline
    std::cout << "\n--- Lancement Kernel 1 (Baseline) ---" << std::endl;
    CUDA_CHECK(hipEventRecord(start));
    Frag_standard_baseline<<<gridDim, blockDim>>>(d_A, d_B, d_out);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Temps: " << milliseconds << " ms" << std::endl;

    CUDA_CHECK(hipMemcpy(h_out.data(), d_out, output_size * sizeof(float), hipMemcpyDeviceToHost));
    
    // Validation
    int errors = 0;
    float epsilon = 1e-2;
    for(int i = 0; i < output_size; ++i) {
        if (std::abs(h_out[i] - ref_out[i]) > epsilon) errors++;
    }
    std::cout << "Validation: " << (errors == 0 ? "PASS" : "FAIL") << " (" << errors << " erreurs)" << std::endl;

    // Benchmark In-Reg
    std::cout << "\n--- Lancement Kernel 2 (In-Register) ---" << std::endl;
    CUDA_CHECK(hipMemset(d_out, 0, output_size * sizeof(float))); 
    CUDA_CHECK(hipEventRecord(start));
    Frag_swapped<<<gridDim, blockDim>>>(d_A, d_B, d_out);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Temps: " << milliseconds << " ms" << std::endl;

    CUDA_CHECK(hipMemcpy(h_out.data(), d_out, output_size * sizeof(float), hipMemcpyDeviceToHost));
    
    errors = 0;
    for(int i = 0; i < output_size; ++i) {
        if (std::abs(h_out[i] - ref_out[i]) > epsilon) errors++;
    }
    std::cout << "Validation: " << (errors == 0 ? "PASS" : "FAIL") << " (" << errors << " erreurs)" << std::endl;
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_out));

    return 0;
}